
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <cmath>

__global__
void mapLog(float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  y[i] = log(x[i]);
}

void cudaMapLog(float *x, int n){
    float *d_x, *d_y;
    // allocate memory on device
    hipMalloc(&d_x, n*sizeof(float));
    hipMalloc(&d_y, n*sizeof(float));
    // copy arrays to devie
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, x, n*sizeof(float), hipMemcpyHostToDevice);

    // find good division
    int division = 2;
    for(;division<=1024; division = division<<1)
      if(n%division) break;

    mapLog<<<n*2/division, division/2>>>(d_x, d_y);

    hipMemcpy(x, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}

int main(void)
{
  // generate randomness
  srand(time(NULL));

  float *h_x;
  int N=1024;

  // allocate memory on host
  h_x = (float*)malloc(N*sizeof(float));

  // list initialization & print
  for (int i = 0; i < N; i++) {
    h_x[i] = rand()/100000;
    printf("%f\n", h_x[i]);
  }

  // map function
  cudaMapLog(h_x, N);

  // show results
  for (int i = 0; i < N; i++) {
    printf("%f\n", h_x[i]);
  }

  free(h_x);
}